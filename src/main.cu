#include "hip/hip_runtime.h"
#include <cstdio>
#include <iostream>
#include <vector>
#include <string>
#include <cstdlib>
#include <climits>
#include <chrono>
#include <ctime>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_texture_types.h>
#include <hip/hip_fp16.h>

#include <opencv2/core.hpp>
#include <opencv2/imgcodecs.hpp>
#include <opencv2/highgui.hpp>
#include <opencv2/opencv.hpp>

#include <GLFW/glfw3.h>

#define PI 3.14159265358979323846f
#define GRAVITATIONAL_CONSTANT 6.674e-3f // real gravitational constant is 6.674e-11
#define RAY_TIME_RESOLUTION 1 // time between two ray positions when computing gravitational lensing
#define RAY_MAX_ITERATIONS 100 // maximum amount of ray positions computed per pixel

void cudaHandleError(hipError_t cudaResult){
    if (cudaResult != hipSuccess) {
        fprintf(stderr, "CUDA handle error: %s: %s\n", hipGetErrorName(cudaResult), hipGetErrorString(cudaResult));
        exit(1);
    }
}

__device__ __host__ float
radians_to_degrees(float radians){
    return radians * 180.0f / PI;
}

__device__ __host__ float
degrees_to_radians(float degrees){
    return degrees * PI / 180.0f;
}

// Function to initialize OpenGL
void initOpenGL(int width, int height) {
    glViewport(0, 0, width, height);
    glMatrixMode(GL_PROJECTION);
    glLoadIdentity();
    glOrtho(0, width, 0, height, -1, 1);
    glMatrixMode(GL_MODELVIEW);
    glLoadIdentity();
    glEnable(GL_TEXTURE_2D);
}

class 
Vector{ // TODO: change to float or half? - maybe template

    public:
        float x;
        float y;
        float z;

        __device__ __host__
        Vector(float x, float y, float z):
            x(x),
            y(y),
            z(z)
        {}

        __device__ __host__ float
        length_2() const{
            return x * x + y * y + z * z;
        }

        __device__ __host__ float
        length() const{
            return std::sqrt(length_2());
        }

        __device__ __host__ Vector
        normalize() const{
            float norm = length();
            if(norm > 0){
                return Vector(x / norm, y / norm, z / norm);
            }
            return Vector(x, y, z);
        }

        __device__ __host__ Vector
        operator+(const Vector& b) const{
            return Vector(x + b.x, y + b.y, z + b.z);
        }

        __device__ __host__ Vector
        operator-(const Vector& b) const{
            return Vector(x - b.x, y - b.y, z - b.z);
        }

        __device__ __host__ float
        operator*(const Vector& b) const{
            return x * b.x + y * b.y + z * b.z;
        }

        __device__ __host__ Vector
        operator^(const Vector& b) const{
            return Vector(y * b.z - z * b.y, z * b.x - x * b.z, x * b.y - y * b.x);
        }

        __device__ __host__ Vector
        operator*(float k) const{
            return Vector(x * k, y * k, z * k);
        }

        __device__ __host__ Vector
        operator/(float k) const{
            return Vector(x / k, y / k, z / k);
        }

        __device__ __host__ Vector
        operator+=(const Vector& b){
            *this = *this + b;
            return *this;
        }

        __device__ __host__ static Vector
        Zero(){
            return Vector(0, 0, 0);
        }

        __device__ __host__ static Vector
        Up(){
            return Vector(0, 1, 0);
        }

        __device__ __host__ static Vector
        Down(){
            return Vector(0, -1, 0);
        }

        __device__ __host__ static Vector
        North(){
            return Vector(1, 0, 0);
        }

        __device__ __host__ static Vector
        South(){
            return Vector(-1, 0, 0);
        }

        __device__ __host__ static Vector
        East(){
            return Vector(0, 0, 1);
        }

        __device__ __host__ static Vector
        West(){
            return Vector(0, 0, -1);
        }
};

class
Color{ // TODO: change to float or half? - maybe template
    public:
        unsigned char red;
        unsigned char green;
        unsigned char blue;
        unsigned char alpha;

        __device__
        Color(  unsigned char red, unsigned char green,
                unsigned char blue, unsigned char alpha):
            red(red),
            green(green),
            blue(blue),
            alpha(alpha)
        {}

        __device__ static Color
        Red(){
            return Color(255, 0, 0, 255);
        }

        __device__ static Color
        Green(){
            return Color(0, 255, 0, 255);
        }

        __device__ static Color
        Blue(){
            return Color(0, 0, 255, 255);
        }

        __device__ static Color
        Black(){
            return Color(0, 0, 0, 255);
        }
};

class
Camera{

    public:
        Vector position;
        Vector direction;
        Vector up;

        float view_plane_distance;
        float view_plane_width;
        float view_plane_height;

        float front_plane_distance;
        float back_plane_distance;

        Camera( Vector position, Vector direction, Vector up,
                float view_plane_distance, float view_plane_width, float view_plane_height,
                float front_plane_distance, float back_plane_distance):

            position(position), direction(direction), up(up),
            view_plane_distance(view_plane_distance), view_plane_width(view_plane_width), view_plane_height(view_plane_height),
            front_plane_distance(front_plane_distance), back_plane_distance(back_plane_distance)
        {}

        Camera( int output_width, int output_height, int field_of_view,
                Vector position, Vector direction, Vector up,
                float front_plane_distance, float back_plane_distance):
            
            position(position), direction(direction), up(up),
            front_plane_distance(front_plane_distance), back_plane_distance(back_plane_distance)
        {
            view_plane_distance = 1;
            float theta = 90.f - float(field_of_view) / 2.f;
            view_plane_width = 2.f * view_plane_distance / std::tan(degrees_to_radians(theta));
            float aspect_ratio = (float)output_width / (float)output_height;
            view_plane_height = view_plane_width / aspect_ratio;
        }
};

class
BlackHole {

    public:
        float mass;
        Vector position;

        BlackHole(float mass, Vector position):
            mass(mass),
            position(position)
        {}
};

__device__ void
set_image_pixel(unsigned char * pixels, int i, int j, int output_width, Color color){
    pixels[i * output_width * 3 + j * 3 + 0] = color.red;
    pixels[i * output_width * 3 + j * 3 + 1] = color.green;
    pixels[i * output_width * 3 + j * 3 + 2] = color.blue;
}

__device__ float
image_to_view_plane(int n, int img_size, float view_plane_size){
    return - n * view_plane_size / img_size + view_plane_size / 2;
}

__device__ __host__ float
angle_between_vectors(Vector a, Vector b){ // result between 0 and 180
    float dot_product = a * b;
    float length_a = a.length();
    float length_b = b.length();

    return radians_to_degrees(std::acos(dot_product / (length_a * length_b)));
}

__device__ Color
extract_texture_color(hipTextureObject_t texture_object, float i, float j, int skybox_width){
    return Color(
        tex2D<float4>(texture_object, j, i).z,
        tex2D<float4>(texture_object, j, i).y,
        tex2D<float4>(texture_object, j, i).x,
        0
    );
}

__global__ void // TODO: recycle variables, use scope operators to conserve register space
ray(
Camera camera,
hipTextureObject_t skybox_texture_object, unsigned char * pixels,
int output_width, int output_height,
int skybox_width, int skybox_height,
int grid_index, int block_size, // currently, even if ray is called as a "remainder" kernel with eg. <<<1920, 128>>>,
                                // block_size is still passed as the block size of a "non-remainder" kernel, eg. 896
                                // in order for the image coordinate arithmetic to be correct
BlackHole * black_holes, int num_black_holes
){
    
    int i = blockIdx.x;
    int j = threadIdx.x + grid_index * block_size;
    
    // initial ray direction
    Vector view_parallel = (camera.up ^ camera.direction).normalize();
    Vector camera_to_view_plane = camera.direction * camera.view_plane_distance;
    float image_to_view_plane_width    = image_to_view_plane(j, output_width , camera.view_plane_width);
    float image_to_view_plane_height   = image_to_view_plane(i, output_height, camera.view_plane_height);
    Vector ray_direction = camera_to_view_plane + view_parallel * image_to_view_plane_width + camera.up * image_to_view_plane_height;

    // gravitational lensing computation
    Vector old_position = camera.position;
    Vector old_velocity = ray_direction; // TODO?: try normalize
    Vector new_position = Vector::Zero();
    Vector new_velocity = Vector::Zero();
    float gravitational_constant = GRAVITATIONAL_CONSTANT;
    float ray_time_resolution = RAY_TIME_RESOLUTION;
                                                /* to paint a pixel black, the ray has to be stuck */
    int escape_sphere_radius = 5;               /* in a sphere of radius = escape_sphere_radius    */
    Vector escape_sphere_center = old_position; /* centered in escape_sphere_center                */
    int escape_sphere_iterations = 10;          /* for escape_sphere_iterations iterations         */                                         
    for(int iter=0; iter<RAY_MAX_ITERATIONS; iter++){
        Vector resultant_force = Vector::Zero();
        for(int black_hole_index=0; black_hole_index<num_black_holes; black_hole_index++){ // TO STUDY?: extract to separate kernel
            Vector black_hole_position = black_holes[black_hole_index].position;
            float black_hole_mass = black_holes[black_hole_index].mass;
            float r_squared = (black_hole_position - old_position).length_2();
            Vector r_hat = (black_hole_position - old_position).normalize();
            resultant_force += r_hat * gravitational_constant * black_hole_mass / r_squared;
        }
        new_velocity = (old_velocity + resultant_force * ray_time_resolution).normalize() * old_velocity.length();
        new_position = old_position + new_velocity * ray_time_resolution;

        if((new_position - escape_sphere_center).length() > escape_sphere_radius){
            // the ray has escaped the sphere, reset the iteration counter and set
            // sphere center to new_position
            escape_sphere_center = new_position;
            escape_sphere_iterations = 10;
        }
        else{
            escape_sphere_iterations--;
            if(escape_sphere_iterations < 0){
                set_image_pixel(pixels, i, j, output_width, Color::Black());
                return;
            }
        }

        old_velocity = new_velocity;
        old_position = new_position;
    }
    ray_direction = new_velocity;
    
    // skybox rendering
    float elevation_angle = angle_between_vectors(Vector::Up(), ray_direction); // 0 degrees <=> straight up, 180 degress <=> straight down
    // project ray_direction on xOz to calculate azimuth
    Vector ray_direction_projection_on_xOz = Vector(ray_direction.x, 0, ray_direction.z); // TODO: reuse ray_direction, dont declare new var
    float projection_north_angle = angle_between_vectors(ray_direction_projection_on_xOz, Vector::North());
    float azimuth_angle = ray_direction_projection_on_xOz.z > 0 ?
                           projection_north_angle :
                    (360 - projection_north_angle); // if z component is negative => azimuth angle > 180 degrees

    // TO STUDY: why did i need printf("") before?

    float skybox_height_coordinate = elevation_angle / 180;
    float skybox_width_coordinate  = azimuth_angle   / 360;

    Color skybox_color = extract_texture_color(skybox_texture_object, skybox_height_coordinate, skybox_width_coordinate, skybox_width);

    set_image_pixel(pixels, i, j, output_width, skybox_color); // TODO: shared mem
}

int
main(int argc, char * argv[]){
/*
args:
1 - output width
2 - output height
3 - field of view (degrees)
4 - skybox filename, optional, must be an equirectangular image (2:1 aspect ratio), default: starmap_2020_2k_gal.png
*/
    int output_width    = atoi(argv[1]);
    int output_height   = atoi(argv[2]);
    int field_of_view   = atoi(argv[3]);
    Vector camera_position  = Vector::Zero();
    Vector camera_direction = Vector::South();
    Vector camera_up        = Vector::Up();
    const float front_plane_distance   = 0;
    const float back_plane_distance    = 1000;
    const float camera_distance_from_center = 60;
    Camera camera(  output_width, output_height, field_of_view,
                    camera_position, camera_direction, camera_up,
                    front_plane_distance, back_plane_distance);

    unsigned char * d_pixels;
    unsigned char * h_pixels = (unsigned char *)malloc(output_width * output_height * 3 * sizeof(unsigned char)); // TODO: check if 3 or 4
    std::vector<unsigned char> h_skybox;
    unsigned int skybox_width   = UINT32_MAX;
    unsigned int skybox_height  = UINT32_MAX;
    std::string skybox_filename = ".\\textures\\";
    BlackHole * d_black_holes;
    std::vector<BlackHole> h_black_holes;
    if(argc > 4){
        skybox_filename += argv[4];
    }
    else{
        skybox_filename += "starmap_2020_2k_gal.png";
    }

    cv::Mat skybox_matrix = cv::imread(skybox_filename);
    if(skybox_matrix.empty()){
        fprintf(stderr, "Error opening skybox file %s\n", skybox_filename.c_str());
        exit(1);
    }
    if(skybox_matrix.channels() == 3){
        cv::cvtColor(skybox_matrix, skybox_matrix, cv::COLOR_BGR2BGRA);
    }
    skybox_matrix.convertTo(skybox_matrix, CV_32FC4);
    printf("Skybox matrix info:\n");
    printf("Datatype code: %d\n", skybox_matrix.type());
    printf("Skybox width: %d\n", skybox_matrix.cols);
    printf("Skybox height: %d\n", skybox_matrix.rows);
    printf("Skybox channels: %d\n", skybox_matrix.channels());

    skybox_width = skybox_matrix.cols;
    skybox_height = skybox_matrix.rows;

    // skybox texture memory
    hipChannelFormatDesc skybox_channel_desc = hipCreateChannelDesc<float4>();
    hipArray_t skybox_array;
    cudaHandleError(hipMallocArray(&skybox_array, &skybox_channel_desc, skybox_width, skybox_height));

    const size_t skybox_source_pitch = skybox_width * sizeof(float4);
    cudaHandleError(hipMemcpy2DToArray(skybox_array, 0, 0, skybox_matrix.data, skybox_source_pitch, skybox_width * sizeof(float4), skybox_height, hipMemcpyHostToDevice));

    hipResourceDesc skybox_resource_desc;
    std::memset(&skybox_resource_desc, 0, sizeof(hipResourceDesc));
    skybox_resource_desc.resType = hipResourceTypeArray;
    skybox_resource_desc.res.array.array = skybox_array;

    hipTextureDesc skybox_texture_desc;
    std::memset(&skybox_texture_desc, 0, sizeof(hipTextureDesc));
    skybox_texture_desc.addressMode[0] = hipAddressModeClamp;
    skybox_texture_desc.addressMode[1] = hipAddressModeClamp;
    skybox_texture_desc.filterMode = hipFilterModeLinear;
    skybox_texture_desc.readMode = hipReadModeElementType;
    skybox_texture_desc.normalizedCoords = true;

    hipTextureObject_t skybox_texture_object = 0;
    cudaHandleError(hipCreateTextureObject(&skybox_texture_object, &skybox_resource_desc, &skybox_texture_desc, nullptr));

    // declaring black holes
    h_black_holes.push_back(BlackHole(500, Vector(0, 0, 0)));
    // h_black_holes.push_back(BlackHole(500, Vector(0, 0, 40)));
    int num_black_holes = (int)h_black_holes.size();

    // allocating device memory
    cudaHandleError(hipMalloc((void **)&d_pixels, output_width * output_height * 3 * sizeof(unsigned char))); // TODO: mallocManaged?
    cudaHandleError(hipMalloc((void **)&d_black_holes, num_black_holes * sizeof(BlackHole))); // TODO: mallocManaged?
    cudaHandleError(hipMemcpy(d_black_holes, h_black_holes.data(), num_black_holes * sizeof(BlackHole), hipMemcpyHostToDevice));

    // initialize GLFW
    GLFWwindow* window;

    if(!glfwInit()){
        return -1;
    }

    window = glfwCreateWindow(640, 480, "Hello World", NULL, NULL);
    if(!window){
        glfwTerminate();
        return -1;
    }

    glfwMakeContextCurrent(window);

    initOpenGL(output_width, output_height);

    GLuint textureID;
    glGenTextures(1, &textureID);
    glBindTexture(GL_TEXTURE_2D, textureID);
    glTexImage2D(GL_TEXTURE_2D, 0, GL_RGB, output_width, output_height, 0, GL_RGB, GL_UNSIGNED_BYTE, h_pixels);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_LINEAR);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_LINEAR);

    float angle = 0;
    float progress_percent = -1;
    auto t_start = std::chrono::high_resolution_clock::now();
    camera.position = Vector(60, 0, 0);

    float black_hole_distance = 40;

    glfwSetWindowMonitor(window, glfwGetPrimaryMonitor(), 0, 0, 1920, 1080, GLFW_DONT_CARE);

    while(angle < 360 && !glfwWindowShouldClose(window)){ // TODO?: call multiple kernels from multiple threads
        // SPIN IN THE CENTER:
        // camera.direction = Vector(std::cos(degrees_to_radians(angle)), 0, std::sin(degrees_to_radians(angle)));
        // ORBIT AROUND CENTER:
        // camera.position = Vector(std::cos(degrees_to_radians(angle)), 0, std::sin(degrees_to_radians(angle))) * camera_distance_from_center;
        // camera.direction = (Vector::Zero() - camera.position).normalize();
        // MOVE INTO CENTER:
        // TODO: issue noticed when camera is at "perfect" integer coordinates, got CUDA error: an illegal memory access was encountered
        camera.position = camera.position + camera.direction / 2;

        int remaining_width = output_width;
        int grid_index = 0;
        constexpr int threads_per_block = 896; // TO STUDY: declared new variables in kernel, tried rendering 1080p
                                     // with 1024 threads per block (what was by defualt) and got:
                                     // "CUDA error: too many resources requested for launch"
                                     // (works with 896 on my GPU, but might differ on others)

        auto frame_start = std::chrono::high_resolution_clock::now();
        while(remaining_width > 0){
            if(remaining_width >= threads_per_block){
                ray<<<output_height, threads_per_block>>>(camera, skybox_texture_object, d_pixels, output_width, output_height, skybox_width, skybox_height, grid_index, threads_per_block, d_black_holes, num_black_holes);
            }
            else{ // remainder kernel call
                ray<<<output_height, remaining_width>>>(camera, skybox_texture_object, d_pixels, output_width, output_height, skybox_width, skybox_height, grid_index, threads_per_block, d_black_holes, num_black_holes);                
            }
            hipError_t hipError_t = hipGetLastError();
            if (hipError_t != hipSuccess) {
                fprintf(stderr, "CUDA error line 468: %s: %s\n", hipGetErrorName(hipError_t), hipGetErrorString(hipError_t));
                exit(1);
            }
            remaining_width -= threads_per_block;
            grid_index++;
        }
        hipDeviceSynchronize();
        auto frame_end = std::chrono::high_resolution_clock::now();
        float frame_duration = std::chrono::duration<float, std::milli>(frame_end - frame_start).count();
        // printf("frame%03d time: %f\n", (int)(angle*2.), frame_duration);
        auto other_start = std::chrono::high_resolution_clock::now();
        hipError_t hipError_t = hipGetLastError();
        if (hipError_t != hipSuccess) {
            fprintf(stderr, "CUDA error line 477: %s: %s\n", hipGetErrorName(hipError_t), hipGetErrorString(hipError_t));
            exit(1);
        }

        cudaHandleError(hipMemcpy(h_pixels, d_pixels, output_width * output_height * 3 * sizeof(unsigned char), hipMemcpyDeviceToHost));

        glBindTexture(GL_TEXTURE_2D, textureID);
        glTexSubImage2D(GL_TEXTURE_2D, 0, 0, 0, output_width, output_height, GL_RGB, GL_UNSIGNED_BYTE, h_pixels);

        glClear(GL_COLOR_BUFFER_BIT);

        glBegin(GL_QUADS);
        glTexCoord2f(0, 0); glVertex2f(0, 0);
        glTexCoord2f(1, 0); glVertex2f((GLfloat)output_width, 0);
        glTexCoord2f(1, 1); glVertex2f((GLfloat)output_width, (GLfloat)output_height);
        glTexCoord2f(0, 1); glVertex2f(0, (GLfloat)output_height);
        glEnd();

        glfwSwapBuffers(window);

        glfwPollEvents();

        // // save to output
        // char * output_path = (char*)malloc(25 * sizeof(char)); //TODO: free
        // sprintf(output_path, "output\\frame%03d.png", (int)(angle*2.));
        // bool imwrite_success = cv::imwrite(output_path, cv::Mat(cv::Size(output_width, output_height), CV_8UC3, h_pixels));
        // if(!imwrite_success){
        //     fprintf(stderr, "Failed to imwrite output frame\n");
        //     exit(1);
        // }

        float current_progress_percent = angle / 360.0f * 100;

        if (current_progress_percent - progress_percent > 1) {
            progress_percent = current_progress_percent;
            // printf("%d%% Done\n", (int)progress_percent);
        }
        
        angle += 0.5;

        // collision
        black_hole_distance-=0.25;
        if(black_hole_distance < 0){
            break;
        }
        h_black_holes.clear();
        h_black_holes.push_back(BlackHole(500, Vector(0, 0, 0)));
        // h_black_holes.push_back(BlackHole(500, Vector(0, 0, black_hole_distance)));
        
        cudaHandleError(hipMemcpy(d_black_holes, h_black_holes.data(), h_black_holes.size() * sizeof(BlackHole), hipMemcpyHostToDevice));
        auto other_end = std::chrono::high_resolution_clock::now();
        float other_duration = std::chrono::duration<float, std::milli>(other_end - other_start).count();
        printf("frame%03d other time: %f\n", (int)(angle*2.), other_duration);
    }
    auto t_end = std::chrono::high_resolution_clock::now();
    float duration = std::chrono::duration<float, std::milli>(t_end - t_start).count();
    printf("Time: %f ms\n", duration);
    printf("Estimated fps: %f\n", 160 / (duration / 1000));

    printf("Done\n");

    glfwTerminate();

    //TODO: free memory
    
    return 0;
}
